#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdio.h>

enum
{
  NELEMS = 1 << 22
};

double wtime()
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

__global__ void vadd(const float *a, const float *b, float *c, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

int main()
{
  size_t size = sizeof(float) * NELEMS;
  double tgpu = 0, tmem = 0;
  
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);

  for (int i = 0; i < NELEMS; ++i)
  {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  float *d_A = NULL, *d_B = NULL, *d_C = NULL;
  tmem = -wtime();
  if (hipMalloc((void **)&d_A, size) != hipSuccess)
  {
    fprintf(stderr, "Allocation error\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **)&d_B, size) != hipSuccess)
  {
    fprintf(stderr, "Allocation error\n");
    exit(EXIT_FAILURE);
  }
  if (hipMalloc((void **)&d_C, size) != hipSuccess)
  {
    fprintf(stderr, "Allocation error\n");
    exit(EXIT_FAILURE);
  }

  if (hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Host to device copying failed\n");
    exit(EXIT_FAILURE);
  }
  if (hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Host to device copying failed\n");
    exit(EXIT_FAILURE);
  }
  tmem += wtime();

  tgpu = -wtime();
  int threadsPerBlock = 1024;
  int blocksPerGrid = (NELEMS + threadsPerBlock - 1) / threadsPerBlock;
  vadd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, NELEMS);
  hipDeviceSynchronize();
  tgpu += wtime();

  if (hipGetLastError() != hipSuccess)
  {
    fprintf(stderr, "Failed to launch kernel!\n");
    exit(EXIT_FAILURE);
  }
  
  tmem -= wtime();
  if (hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Device to host copying failed\n");
    exit(EXIT_FAILURE);
  }
  tmem += wtime();

  for (int i = 0; i < NELEMS; ++i)
  {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }
  
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
  hipDeviceReset();

  printf("GPU time (sec.): %.6f\n", tgpu);
  printf("Memory ops. (sec.): %.6f\n", tmem);
  printf("Total time (sec.): %.6f\n", tgpu + tmem);

  return 0;
}
